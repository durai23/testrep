// include libraries
#include <stdio.h>
#include <math.h>
#include <omp.h>
#include "hipblas.h"
#include "hip/hip_runtime.h"

#define nstreams 4    

int main () {

  // banner
  printf ("\n\n     GPU Computing Advanced Workshop Exercise\n");
  printf (    "     ==========================================\n");
  printf (  "\n     Tiled Matrix-Matrix Multiplication\n");
  printf (    "     Using NVIDIA cuBLAS Library with Streams\n");

  // echo device data
  int idevice = 0;
  hipSetDevice(idevice);
  hipDeviceProp_t dprops;
  hipGetDeviceProperties( &dprops, idevice );
  printf ("\n     Device name = %s, with compute capability %d.%d \n", 
	  dprops.name, dprops.major, dprops.minor);

  // define parameters
  int n = 1024;   // matrix dimension - all matrices being multiplied will be square
  int m = 512;    // tile size - tiles will be square, n must be divisible by m
  printf ("\n     Matrix sizes: %d x %d, tile size: %d x %d\n", n,n,m,m);
  
  // allocate arrays
  double *a;
  double *b;
  double *c;
  a = (double *) malloc ( n*n*sizeof(double) );
  b = (double *) malloc ( n*n*sizeof(double) );
  c = (double *) malloc ( n*n*sizeof(double) );
  
  // initialize data
  #pragma omp parallel for
  for ( int row = 0; row<n; row++ ) {
    for ( int col = 0; col<n; col++ ) {
      // data in row-major format
      a[row*n+col] = sin( 0.01*col ) + cos( 0.013*row );
      b[row*n+col] = sin( 0.017*col ) + cos( 0.03*row );
      c[row*n+col] = 0.0;
    }
  }

  // create communcations arrays
  double *pa;
  double *pb;
  double *pc;
  hipHostMalloc ( &pa, m*m*sizeof(double)*nstreams , hipHostMallocDefault);
  hipHostMalloc ( &pb, m*m*sizeof(double)*nstreams , hipHostMallocDefault);
  hipHostMalloc ( &pc, m*m*sizeof(double)*nstreams , hipHostMallocDefault);
	  
  // create a handle to cuBlas
  hipblasHandle_t cublasHandle;
  hipblasCreate( &cublasHandle );

  // allocate space on device - 3 tiles for a, b, c
  double *d_a;
  double *d_b;
  double *d_c;
  hipMalloc ( &d_a, nstreams*m*m*sizeof(double) );
  hipMalloc ( &d_b, nstreams*m*m*sizeof(double) );
  hipMalloc ( &d_c, nstreams*m*m*sizeof(double) );

  int offset = m*m;
  int ntiles = n/m;

  hipStream_t myStreams[nstreams];
  for ( int i=0; i<nstreams; i++ ) {
    hipStreamCreate( &myStreams[i] );
  }

  hipEvent_t bufferfilled[nstreams];
  for ( int i=0; i<nstreams; i++ ) {
    hipEventCreate ( &bufferfilled[i] );
  }

  // record start time
  hipEvent_t t_start;
  hipEvent_t t_end;
  hipEventCreate (&t_start);
  hipEventCreate (&t_end);
  hipEventRecord (t_start,0);

  // caches for indices of previous tiles in streams
  int prowtile[nstreams];
  int pcoltile[nstreams];

  // PERFORM MULTIPLICATION
  {

    double alpha = 1.0;
    double beta = 1.0; 

    int ibuff = 0;
    int itile = 0;

    // loop over inner tile dimension
    for ( int iktile = 0; iktile < ntiles; iktile++ ) {
  
      // loop over row tiles
      for ( int irowtile = 0; irowtile < ntiles; irowtile++ ) {

        // loop over column tiles
        for ( int icoltile = 0; icoltile < ntiles; icoltile++ ) {
	  
	  if ( itile >= nstreams ) {

	    // block the host until this streams buffers are available
	    // (that is, all previous operations in this stream have completed)
	    hipEventSynchronize ( bufferfilled[ibuff] );

	    // copy result in pinned buffer back to global matrix
            # pragma omp parallel for
	    for ( int i=0; i<m; i++ ) {
	      for ( int j=0; j<m; j++ ) {
		c[(prowtile[ibuff]*m+i)*n+pcoltile[ibuff]*m+j] = pc[ibuff*offset+i*m+j];
	      }
	    }
	  } 

	  // copy next tile to pinned buffer
          # pragma omp parallel for
	  for ( int i=0; i<m; i++ ) {
	    for ( int j=0; j<m; j++ ) {
	      pa[ibuff*offset+i*m+j] = a[(irowtile*m+i)*n+iktile*m+j];
	      pb[ibuff*offset+i*m+j] = b[(iktile*m+i)*n+icoltile*m+j];
	      pc[ibuff*offset+i*m+j] = c[(irowtile*m+i)*n+icoltile*m+j];
	    }
	  }

	  // copy tile data to device
	  hipMemcpyAsync ( &d_a[ibuff*offset], &pa[ibuff*offset], m*m*sizeof(double), hipMemcpyHostToDevice, myStreams[ibuff] );
	  hipMemcpyAsync ( &d_b[ibuff*offset], &pb[ibuff*offset], m*m*sizeof(double), hipMemcpyHostToDevice, myStreams[ibuff] );
	  hipMemcpyAsync ( &d_c[ibuff*offset], &pc[ibuff*offset], m*m*sizeof(double), hipMemcpyHostToDevice, myStreams[ibuff] );

	  // tell cuBLAS which stream to use
	  hipblasSetStream( cublasHandle, myStreams[ibuff] );

	  // perform dgemm
	  hipblasDgemm ( cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, m, m, &alpha, &d_a[ibuff*offset], m, &d_b[ibuff*offset], m, &beta, &d_c[ibuff*offset], m ); 
	  prowtile[ibuff] = irowtile;
	  pcoltile[ibuff] = icoltile;

	  // copy result back to host
	  hipMemcpyAsync ( &pc[ibuff*offset], &d_c[ibuff*offset], m*m*sizeof(double), hipMemcpyDeviceToHost, myStreams[ibuff] );

	  // this event will signal when the D2H copy of the result has completed
	  hipEventRecord ( bufferfilled[ibuff], myStreams[ibuff] );
	  
	  // update buffer / stream
	  ibuff++;
	  ibuff = ibuff%nstreams;
	  itile++;

	}
      }
    }

    for ( itile=0; itile < nstreams; itile ++ ) {

      // make sure that buffers are free
      hipStreamSynchronize ( myStreams[itile] );

      // copy result in pinned buffer back to source 
      # pragma omp parallel for
      for ( int i=0; i<m; i++ ) {
	for ( int j=0; j<m; j++ ) {
	  c[(prowtile[itile]*m+i)*n+pcoltile[itile]*m+j] = pc[itile*offset+i*m+j];
	}
      }
	    
    }

  }

  // record end time
  hipEventRecord (t_end,0);
  hipEventSynchronize(t_end);
  float et;
  hipEventElapsedTime (&et, t_start, t_end);
    
  // report results
  printf("\n     reference (768,768) = %4.4f ", c[768*n+768] );
  printf("\n     elapsedTime        = %4.4f seconds\n", (double)et/1000.);     // cudaEventElapsedTime is in milliseconds
  printf(  "     gigaflops achieved = %4.4f Gflops/s\n\n\n", 2.0e-6*n*n*n/et); // 2( * and + ) *n (inner dimension)*n^2(result size)/(time in ms.)

  // clean up
  hipblasDestroy ( cublasHandle );
  hipEventDestroy ( t_start  );
  hipEventDestroy ( t_end );

  hipHostFree ( pa );
  hipHostFree ( pb );
  hipHostFree ( pc );

  hipFree ( d_a );
  hipFree ( d_b );
  hipFree ( d_c );

}
